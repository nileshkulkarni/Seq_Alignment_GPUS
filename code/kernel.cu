#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <strstream>
#include <iostream>

using namespace std;
#define gap_penalty 3  //penalty to substitute with a gap //insertion or deletion
#define sub_penalty 5  //penalty to substitute with a different character

#define NO_BLOCKS  1024
#define NO_THREADS_PER_BLOCK 512 //kept as a multiple of 32 so as to make sure we make use of optimum number of warps

int findAlignment(int *outarr , char *outstr1 , char *outstr2, char *str1 , char *str2 , int l1 , int l2){
	int y = l1; //tracks the current index for string 1 i.e at any time l1 - y characters of str1 have been matched 
	int x = l2; //tracks the current index for string 2 i.e at any time l1 - x characters of str2 have been matched 
	//outarr is the matrix with alignment values inserted
	//outstr1 and outsrt2 are the final matched strings
	outstr1[l1+l2] = '\0'; 
	outstr2[l1+l2] = '\0';

	int val1,val2,val3;
	int t = l1+l2-1;
	while(y>0 && x>0){
		val1 = outarr[y*(l2+1) + x-1] +  gap_penalty;
        val2 = outarr[(y-1)*(l2+1) + x] + gap_penalty;
		val3 = outarr[(y-1)*(l2+1) + x-1] + ((str1[y-1] != str2[x-1]) * sub_penalty);	
		
		//
		if(outarr[y*(l2+1) + x] == val1){
			outstr2[t] = str2[x-1];
			outstr1[t--] = '_';  //blank
			x--;
			continue;
		}
		
		if(outarr[y*(l2+1) + x] == val2){
			outstr1[t] = str1[y-1];
			outstr2[t--] = '_';  //blank
			y--;
			continue;
		}

		if(outarr[y*(l2+1) + x] == val3){
			outstr2[t] = str2[x-1];
			outstr1[t--] = str1[y-1];  //substitute
			x--; y--;
			continue;
		}
	}

	//substitute the remaining elements with _ and other with elements as that of the input string
	for(int i=x;i>0 ; i--,t--){
		outstr2[t] = str2[i-1];
		outstr1[t] = '_';
	}

	for(int j=y;j>0 ; j--,t--){
		outstr1[t] = str1[j-1];
		outstr2[t] = '_';
	}

	//outstr1 = outstr1 + t;
	//outstr2 = outstr2 + t; 
	return t+1;
}





//device function for max of 3 numbers written avoiding many conditional statements.
__device__ int  mymax(int a ,int  b,int  c ){
	int max =a;
	max = (max<b)*b + (max>=b)*max;
	max = (max<c)*c + (max>=c)*max;
	return max;
}

//device function for min of 3 numbers written avoiding many conditional statements.
__device__ int  mymin3(int a ,int  b,int  c ){
	int min =a;
	min = (min>b)*b + (min<=b)*min;
	min = (min>c)*c + (min<=c)*min;
	return min;
}


//min of 2 nunmbers, host function
int  mymin(int a ,int  b){
	
	return (a>=b)*b + (a<b)*a;

}


//prints the input vector
void print_vector(int *arr ,  int len){

	for(int i=0; i<len;i++){
		printf("%d , ",arr[i]);
	}
	printf("\n");
}


__global__ void dpf(char *str1 , char *str2 , int *out_arr, int p, int q,int curr_x,int curr_y)
{
	int id = blockDim.x* blockIdx.x + threadIdx.x;
	

/*
	|
	|					diagonal_x   \				
	| diagonal_y           --------- |
	|								 /
   \_/ 

*/


	int diagonal_x ,diagonal_y;
		
	
//	while((curr_y != p) || (curr_x != q+1)){
		diagonal_x = curr_x+ id;
		diagonal_y = curr_y-id;

		if(diagonal_x <= q && diagonal_y>=0){

			out_arr[diagonal_y*(q+1) + diagonal_x] = (diagonal_x==0 && diagonal_y ==0)*0	 
												+(diagonal_x==0 && diagonal_y !=0)*(diagonal_y * gap_penalty)
												+(diagonal_x !=0 && diagonal_y == 0)*(diagonal_x * gap_penalty)
												+(diagonal_x !=0 && diagonal_y !=0)*mymin3(out_arr[diagonal_y*(q + 1) + diagonal_x -1] +  gap_penalty,
																						out_arr[(diagonal_y-1)*(q + 1)+ diagonal_x] + gap_penalty,
																						out_arr[(diagonal_y-1)*(q + 1)  + diagonal_x - 1] + 
																						        (str1[diagonal_y-1] != str2[diagonal_x-1]) * sub_penalty);	
			}

		//curr_x = curr_x + ((curr_y/p) * 1);
		//curr_y = mymin((curr_y +1),p);  		
		//__syncthreads();	
	//}
		

}
	

hipError_t launchProg(char *str1 ,char *str2, int* outarr , int p, int q){
	// Steps in cuda program:
	// allocate variables space on the cudamemory
	// copy the data
	// call the kernel function

	char *str1_k;
	char *str2_k;
	int *out_k; //output 2d array
	printf("1\n");
	
	hipError_t cudaStatus = hipSetDevice(0);
    



	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error1;
    }

	// Allocate GPU buffers for three vectors (two input, one output)    .
	
printf("2\n");
	cudaStatus = hipMalloc((void**)&out_k, (p+1)*(q+1) * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stdout, "hipMalloc failed! , could not allot space to output array");
			//fprintf(stdout, "%s" , hipGetErrorString(cudaStatus));
			goto Error1;
		}	



		cudaStatus = hipMalloc((void**)&str1_k, p * sizeof(char));
		if (cudaStatus != hipSuccess) {
			fprintf(stdout, "hipMalloc failed!");
			goto Error1;
		}


			cudaStatus = hipMalloc((void**)&str2_k, q *  sizeof(char));
		if (cudaStatus != hipSuccess) {
			fprintf(stdout, "hipMalloc failed!");
			goto Error1;
		}
	

		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(str1_k,str1, p * sizeof(char), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stdout, "hipMemcpy failed!");
			goto Error1;
		}

		cudaStatus = hipMemcpy(str2_k,str2, q * sizeof(char), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stdout, "hipMemcpy failed!");
			goto Error1;
		}
	
		// Launch a kernel on the GPU with one thread for each element.
		/*
	|
	|				q	   \				
	| p           --------- |
	|					   /
   \_/ 

*/

		int curr_x =0, curr_y=0;
		while((curr_y != p) || (curr_x != q+1)){
			dpf<<<NO_BLOCKS, NO_THREADS_PER_BLOCK>>>(str1_k , str2_k ,out_k,p,q,curr_x,curr_y);
			curr_x = curr_x + ((curr_y/p) * 1);
			curr_y = mymin((curr_y +1),p);  			
		}

		
		
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error1;
		}
    
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error1;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(outarr, out_k, (p + 1)*(q + 1)* sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy of output array failed!");
			goto Error1;
		}

Error1:
    hipFree(out_k);
    return cudaStatus;
}

















int main() {
   // int n = 25;
//	char seq1[30000],seq2[30000];


	char *seq1 = "aishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfsjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfa";
	char *seq2=  "SailAwayFromTheShoresSailAwayFromTheShoresSailAwayFromTheShoressljflkajdlkjalkdsjflkjdfdsjkdfaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkhwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkhwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljflaishwariyaVipulajsljflkhwariyaVipulajsljflkajdlkjalkdsjflkjdfdaishwariyaVipulajsljflkajdlkjalkdsjflkjdfdsjkdfjjlsajfkvipualharssljfjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfasjflkjdfdsjkdfjjlsajfkvipualharssljfa";

	int size;
	ifstream file;
/*	
	file.open("data.txt",ios::in);

	if(file.is_open()){
		file.getline(seq1, 30000);
		file.getline(seq2,30000);
	//	printf("Input string 1 is %s\n %s ", seq1 , seq2);
		file.close();
	
	}
*/

//	else printf("file could not be opened");

	char *str1 = seq1;
    char *str2 = seq2;
	//char *str1 = "aishwariyaAbhiVipul";
	//char *str2 = "aishwariyaVipul";
	int l1 = strlen(str1);
	int l2 = strlen(str2);
	char *outstr1  = new char[l1+l2+1];
	char *outstr2  = new char[l2+l2+1];
	int *outarr = new int[(l1+1)*(l2+1)];
    // Add vectors in parallel.
    hipError_t cudaStatus  = launchProg(str1 , str2 , outarr , l1 , l2);
	
		
	if (cudaStatus != hipSuccess) {
        fprintf(stderr, "launchProg failed!");
        return 1;
    }


	int offset = findAlignment(outarr , outstr1 , outstr2 , str1 , str2 ,l1,l2);
	printf("Aligned Strings are : \n %s \n %s \n" , outstr1 + offset, 
									outstr2+offset);


//output the table
/*   
	for(int i=0;i<=l1;i++){
	   for(int j=0;j<=l2;j++){
		   printf("%d " , outarr[i*(l2+1) + j]);
	   }
	   printf("\n");
   }
*/	


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
 return 0;
}


